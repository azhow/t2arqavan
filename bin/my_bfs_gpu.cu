
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stack>
#include <queue>
#include <chrono>
#include <ctime>  
#include <stdlib.h>

using namespace std;

#define INF 99999
#define GOAL 5000

__global__ void vecBfs(int nNodes, int** graph, bool *visited, bool *done)
{
	for(int v = blockIdx.x; v < nNodes; ++v){
            //cout << "visited " << v << endl;
            if(v == GOAL)
            {
                //cout << "Found " << GOAL << endl;
                *done = true;
            }
            for(int i = blockIdx.y; i < nNodes; ++i){
                if(graph[v][i] != INF && v != i){
                    if(visited[i] == false)
                    {
                        visited[i] = true;
                        *done = false;
                    }
                }
            }
	}
}

int main(int argc, char **argv){
	int** graph;
    int a, b, w, nNodes;

    if (argc > 1)
    {
        ifstream inputfile(argv[1]);
        inputfile >> nNodes;
        graph = new int*[nNodes];
        for (int i = 0; i < nNodes; ++i)
        {
            graph[i] = new int[nNodes]; 
            for (int j = 0; j < nNodes; ++j)
                graph[i][j] = INF;
        }
        while (inputfile >> a >> b >> w)
        {
            graph[a][b] = w;
            graph[b][a] = w;
        }
    }
	
	bool *visited = new bool[nNodes];
	/* BFS */
	for(int i = 0; i < nNodes; i++)
		visited[i] = false;

   	queue<int> q;
	q.push(0);
	visited[0] = true;
	
	/* GPU Setup */
	int indx = 0, blockSize = 32;
	int numBlocks = (nNodes + blockSize - 1) / blockSize;
    bool* device_visited;
    bool* device_done;
    bool* done;
	*done = true;
    hipMalloc(&device_visited, nNodes * sizeof(bool));
    hipMemcpy(device_visited, visited, nNodes * sizeof(bool),hipMemcpyHostToDevice);
    hipMalloc(&device_done, sizeof(bool));
    hipMemcpy(device_done, done, nNodes * sizeof(bool),hipMemcpyHostToDevice);

    /* BFS */
    auto start = std::chrono::system_clock::now();
	while(!(*done)){
        *done = false;
        /* For all V in Graph - total of nNodes V's*/
        vecBfs<<<numBlocks,blockSize>>>(nNodes, graph, device_visited, device_done);
        hipDeviceSynchronize();
        *done = *device_done;
    }
    /* BFS DONE */

	auto end = std::chrono::system_clock::now();
	auto timeElapsed = (end - start);
	auto sec = std::chrono::duration_cast<std::chrono::seconds>(timeElapsed).count();
	for(int i = 0; i < nNodes; i++)
		free(graph[i]);
	free(visited);
}

