
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stack>
#include <queue>
#include <chrono>
#include <ctime>  
#include <stdlib.h>

using namespace std;

#define INF 99999
#define GOAL 5000

__global__ void vecBfs(int nNodes, int** graph, bool *visited, bool *notdone)
{
	for(int v = blockIdx.x * blockDim.x + threadIdx.x; v < nNodes; v+= blockDim.x * gridDim.x){
            //printf("visited %d\n",v);
            if(v == GOAL)
            {
                //cout << "Found " << GOAL << endl;
                *notdone = true;
            }
            for(int i = blockIdx.y * blockDim.y + threadIdx.y; i < nNodes; i += blockDim.y * gridDim.y){
                if(graph[v][i] != INF && v != i){
                    if(visited[i] == false)
                    {
                        visited[i] = true;
                        *notdone = false;
                    }
                }
            }
	}
}

int main(int argc, char **argv){
	int** graph;
    int a, b, w, nNodes;

    if (argc > 1)
    {
        ifstream inputfile(argv[1]);
        inputfile >> nNodes;
        graph = new int*[nNodes];
        for (int i = 0; i < nNodes; ++i)
        {
            graph[i] = new int[nNodes]; 
            for (int j = 0; j < nNodes; ++j)
                graph[i][j] = INF;
        }
        while (inputfile >> a >> b >> w)
        {
            graph[a][b] = w;
            graph[b][a] = w;
        }
    }
	
	bool *visited = new bool[nNodes];
	/* BFS */
	for(int i = 0; i < nNodes; i++)
		visited[i] = false;

   	queue<int> q;
	q.push(0);
	visited[0] = true;
	/* GPU Setup */
	int blockSize = 32;
	int numBlocks = (nNodes + blockSize - 1) / blockSize;
    bool* device_visited;
    bool* device_notdone;
    bool notdone;
    notdone = true;
    hipMalloc(&device_visited, nNodes * sizeof(bool));
    hipMemcpy(device_visited, visited, nNodes * sizeof(bool),hipMemcpyHostToDevice);
    hipMalloc(&device_notdone, sizeof(bool));
    hipMemcpy(device_notdone, &notdone, nNodes * sizeof(bool),hipMemcpyHostToDevice);
    /* BFS */
    auto start = std::chrono::system_clock::now();
	while(notdone){
        notdone = false;
        /* For all V in Graph - total of nNodes V's*/
        vecBfs<<<numBlocks,blockSize>>>(nNodes, graph, device_visited, device_notdone);
        hipDeviceSynchronize();
        hipMemcpy(&notdone, device_notdone, sizeof(bool),hipMemcpyDeviceToHost);
    }
    /* BFS DONE */

	auto end = std::chrono::system_clock::now();
	auto timeElapsed = (end - start);
	auto sec = std::chrono::duration_cast<std::chrono::milliseconds>(timeElapsed).count();
	cout << sec << "\n";
	for(int i = 0; i < nNodes; i++)
		free(graph[i]);
	free(visited);
}

