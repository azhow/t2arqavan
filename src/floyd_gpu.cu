#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
// C++ Program for Floyd Warshall Algorithm  
//#include <bits/stdc++.h> 
#include <chrono>
#include <ctime>  
using namespace std; 
  
/* Define Infinite as a large enough 
value.This value will be used for  
vertices not connected to each other */
#define INF 99999  
  
// Solves the all-pairs shortest path  
// problem using Floyd Warshall algorithm  
__global__ void vecFloydWarshall(int** graph, int* dist, int nNodes, int k)
{
	/* Add all vertices one by one to
	the set of intermediate vertices.
	---> Before start of an iteration,
	we have shortest distances between all
	pairs of vertices such that the
	shortest distances consider only the
	vertices in set {0, 1, 2, .. k-1} as
	intermediate vertices.
	----> After the end of an iteration,
	vertex no. k is added to the set of
	intermediate vertices and the set becomes {0, 1, 2, .. k} */
	// Pick all vertices as source one by one  
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += blockDim.x * gridDim.x)
	{
		for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < nNodes; j += blockDim.y * gridDim.y)
		{
			// Pick all vertices as destination for the  
			// above picked source  
			// If vertex k is on the shortest path from  
			// i to j, then update the value of dist[i][j]  
			//printf("j: %d\n",j);
			if (dist[i*nNodes+k] + dist[k*nNodes+j] < dist[i*nNodes+j]){
				dist[i*nNodes+j] = dist[i*nNodes+k] + dist[k*nNodes+j];
				//printf("i %d, j %d\n",i,j);
			}
		}
	}
}

/* A utility function to print solution */
void printSolution(int** dist, int nNodes)
{
	for (int i = 0; i < nNodes; i++)
	{
		for (int j = 0; j < nNodes; j++)
		{
			if (dist[i][j] == INF)
				cout << "INF" << "     ";
			else
				cout << dist[i][j] << "     ";
		}
		cout << endl;
	}
}

// This code is contributed by rathbhupendra 

int main(int argc, char **argv){
	int** graph;
	int** dist;
	int a, b, w, nNodes;
	int* device_dist;
	int* aux_dist;

	if (argc > 1) 
	{
		cout << "input file is " << argv[1] << endl;
		ifstream inputfile(argv[1]);
		inputfile >> nNodes;
		graph = new int*[nNodes];
		for (int i = 0; i < nNodes; ++i)
		{
			graph[i] = new int[nNodes];
			for (int j = 0; j < nNodes; ++j)
				graph[i][j] = INF;	
		}
		while (inputfile >> a >> b >> w)	
		{
			graph[a][b] = w;
			graph[b][a] = w;
		}
	}
  
	dist = new int*[nNodes];
	aux_dist = new int[nNodes * nNodes];
	for (int i = 0; i < nNodes; ++i)
		dist[i] = new int[nNodes];
    /* dist[][] will be the output matrix  
    that will finally have the shortest  
    distances between every pair of vertices */
    int  i, j, k;  
  
    /* Initialize the solution matrix same  
    as input graph matrix. Or we can say  
    the initial values of shortest distances 
    are based on shortest paths considering  
    no intermediate vertex. */
    for (i = 0; i < nNodes; i++)  {
        for (j = 0; j < nNodes; j++) {
			dist[i][j] = graph[i][j];  
			aux_dist[i*nNodes+j] = graph[i][j];
		}
	}
	//cout << graph[0][1] << " vs " << aux_dist[1] << "at position (" << 0 << "," << 1 << ")\n";
	
	hipMalloc(&device_dist, nNodes * nNodes * sizeof(int));
	hipMemcpy(device_dist, aux_dist, nNodes * nNodes * sizeof(int),hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (nNodes + blockSize - 1) / blockSize;
	auto start = std::chrono::system_clock::now();
	for (int k = 0; k < nNodes; ++k){
		vecFloydWarshall<<<numBlocks, blockSize>>>(graph, device_dist, nNodes, k);
		hipDeviceSynchronize();	
		//cout << "currently in " << k << endl;
	}
	auto end = std::chrono::system_clock::now();
	auto timeElapsed = (end - start);
	hipMemcpy(aux_dist, device_dist, nNodes * nNodes * sizeof(int),hipMemcpyDeviceToHost);

    for (i = 0; i < nNodes; i++)  {
        for (j = 0; j < nNodes; j++) {
			dist[i][j] = aux_dist[i*nNodes+j];
		}
	}
	//cout << graph[0][1] << " vs " << aux_dist[1] << "at position (" << 0 << "," << 1 << ")\n";
    // Print the shortest distance matrix  
    //printSolution(dist, nNodes);  
	auto sec = std::chrono::duration_cast<std::chrono::seconds>(timeElapsed).count();
	cout << "Computation time: " << sec << "\n";


    return 0;  
}  
  
